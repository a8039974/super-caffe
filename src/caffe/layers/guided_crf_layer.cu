#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

//#include "thrust/device_vector.h"

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/layers/guided_crf_layer.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
template <typename Dtype>
static __global__ void softmax_forward_kernel(const int maxStates,const int nNodes, const Dtype * energy,Dtype * prob)
{
	CUDA_KERNEL_LOOP(n, nNodes)
	{
		for(int s=0;s<maxStates;s++)
			prob[s*nNodes+n] = energy[s*nNodes+n];

		Dtype max_prob = Dtype(-FLT_MAX);
		for(int s=0;s<maxStates;s++)
			max_prob =max(max_prob,prob[s*nNodes+n]);

		for(int s=0;s<maxStates;s++)
			prob[s*nNodes+n] -= max_prob;

		Dtype sum = 0;
		for(int s=0;s<maxStates;s++)
			sum += exp(prob[s*nNodes+n]);

		for(int s=0;s<maxStates;s++)
			prob[s*nNodes+n] = exp(prob[s*nNodes+n]) / sum;
	}
}
template <typename Dtype>
static __global__ void softmax_backward_kernel(const int maxStates,const int nNodes, const Dtype * top_diff,const Dtype *prob,Dtype * bottom_diff)
{
	CUDA_KERNEL_LOOP(ind, nNodes*maxStates)
	{
		int n=ind % nNodes;
		int s=ind / nNodes;
		Dtype sum = 0;
		for(int s2=0;s2<maxStates;s2++)
			 sum += top_diff[s2*nNodes+n]*prob[s2*nNodes+n]*(Dtype(s==s2)-prob[s*nNodes+n]);
		bottom_diff[s*nNodes+n] = sum;
	}
}
//--------------------------------------------------------------
template <typename Dtype>
static __global__ void vector_product_kernel(const int num,const int channels1,const int channels2, const int spatial_dim,const Dtype * a,const Dtype * b,Dtype *var)//var = a .* b
{
	CUDA_KERNEL_LOOP(ind, spatial_dim*channels1*channels2*num)
	{	
		int n   = ind / spatial_dim / channels1 / channels2;
		int c2  = ind / spatial_dim / channels1 % channels2;
		int c1  = ind / spatial_dim % channels1;
		int s   = ind % spatial_dim;
		
		
		var[ind]=a[(n*channels1+c1)*spatial_dim+s]*b[(n*channels2+c2)*spatial_dim+s];
	}
}
template <typename Dtype>
static __global__ void substract_vector_product_kernel(const int num, const int channels1,const int channels2,const int spatial_dim,const Dtype *avg,const Dtype *a,const Dtype *b, Dtype * var)//var = avg - a.*b;
{
	CUDA_KERNEL_LOOP(ind, spatial_dim*channels1*channels2*num)
	{
		int n  = ind / spatial_dim / channels1 / channels2;
		int c2 = ind / spatial_dim / channels1 % channels2;
		int c1 = ind / spatial_dim % channels1;	
		int s  = ind % spatial_dim;
		var[ind]=avg[ind]-a[(n*channels1+c1)*spatial_dim+s]*b[(n*channels2+c2)*spatial_dim+s];
	}
}
template <typename Dtype>
static __global__ void inv_var_I_eps_kernel_3(const int num, const int channels, const int spatial_dim, const Dtype eps,Dtype *var_I,Dtype *inv_var_I)
{
	CUDA_KERNEL_LOOP(ind, spatial_dim*num)
	{
		int n = ind / spatial_dim;
		int s = ind % spatial_dim;
		
		for(int c=0;c<channels;c++)
			var_I[(n*channels*channels+(c*channels+c))*spatial_dim+s]=var_I[(n*channels*channels+(c*channels+c))*spatial_dim+s]+eps;

		Dtype det = var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]*(var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]-var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s])
				- var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]*(var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]-var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s])
				+ var_I[(n*channels*channels+0*channels+2)*spatial_dim+s]*(var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]-var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]);

		inv_var_I[(n*channels*channels+0*channels+0)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+0*channels+1)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+0*channels+2)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]
				-var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]);

		inv_var_I[(n*channels*channels+1*channels+0)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+0*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+1*channels+1)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+0*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+1*channels+2)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]
				-var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]);

		inv_var_I[(n*channels*channels+2*channels+0)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+0*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+2*channels+1)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+0*channels+2)*spatial_dim+s]
				-var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+2)*spatial_dim+s]);
		inv_var_I[(n*channels*channels+2*channels+2)*spatial_dim+s] = 1/det*(var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]
				-var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]*var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]);
	}

}
template <typename Dtype>
static __global__ void div_sum_kernel_3(const int num, const int channels,const int maxStates,const int spatial_dim,const Dtype *inv_var_I,const Dtype *cov_Ip,
																 Dtype *a)
{
	CUDA_KERNEL_LOOP(ind, spatial_dim*maxStates*num)
	{
		int n = ind / spatial_dim / maxStates;
		int m = ind / spatial_dim % maxStates;
		int s = ind % spatial_dim;
		
		
		a[((n*maxStates+m)*channels+0)*spatial_dim+s] = cov_Ip[((n*maxStates+m)*channels+0)*spatial_dim+s]*inv_var_I[(n*channels*channels+0*channels+0)*spatial_dim+s]
																		+ cov_Ip[((n*maxStates+m)*channels+1)*spatial_dim+s]*inv_var_I[(n*channels*channels+0*channels+1)*spatial_dim+s]
																		+ cov_Ip[((n*maxStates+m)*channels+2)*spatial_dim+s]*inv_var_I[(n*channels*channels+0*channels+2)*spatial_dim+s];

		a[((n*maxStates+m)*channels+1)*spatial_dim+s]	= cov_Ip[((n*maxStates+m)*channels+0)*spatial_dim+s]*inv_var_I[(n*channels*channels+1*channels+0)*spatial_dim+s]
																	  + cov_Ip[((n*maxStates+m)*channels+1)*spatial_dim+s]*inv_var_I[(n*channels*channels+1*channels+1)*spatial_dim+s]
																		+ cov_Ip[((n*maxStates+m)*channels+2)*spatial_dim+s]*inv_var_I[(n*channels*channels+1*channels+2)*spatial_dim+s];

		a[((n*maxStates+m)*channels+2)*spatial_dim+s] = cov_Ip[((n*maxStates+m)*channels+0)*spatial_dim+s]*inv_var_I[(n*channels*channels+2*channels+0)*spatial_dim+s]
																		+ cov_Ip[((n*maxStates+m)*channels+1)*spatial_dim+s]*inv_var_I[(n*channels*channels+2*channels+1)*spatial_dim+s]
																		+ cov_Ip[((n*maxStates+m)*channels+2)*spatial_dim+s]*inv_var_I[(n*channels*channels+2*channels+2)*spatial_dim+s];
	}
}
template <typename Dtype>
static __global__ void substract_vector_matrix_product_kernel_3(const int num, const int channels,const int maxStates,const int spatial_dim,const Dtype * mean_p,const Dtype * a,const Dtype * mean_I,Dtype *b)//	b = mean_p - mean_I *. a;
{
	CUDA_KERNEL_LOOP(ind, spatial_dim*maxStates*num)
	{
		int n = ind / spatial_dim / maxStates;
		int m = ind / spatial_dim % maxStates;
		int s = ind % spatial_dim;
		
		b[ind] = mean_p[ind]
				   - mean_I[(n*3+0)*spatial_dim+s] * a[((n*maxStates+m)*channels+0)*spatial_dim+s]
				   - mean_I[(n*3+1)*spatial_dim+s] * a[((n*maxStates+m)*channels+1)*spatial_dim+s]
				   - mean_I[(n*3+2)*spatial_dim+s] * a[((n*maxStates+m)*channels+2)*spatial_dim+s];
	}
}
template <typename Dtype>
static __global__ void vector_matrix_product_sum_kernel_3(const int num, const int channels,const int maxStates,const int spatial_dim,const Dtype *mean_a,const Dtype *I,const Dtype *mean_b,Dtype *q)// q = I .* mean_a + mean_b;
{

	CUDA_KERNEL_LOOP(ind, spatial_dim*maxStates*num)
	{
		int n = ind / spatial_dim / maxStates;
		int m = ind / spatial_dim % maxStates;
		int s = ind % spatial_dim;
		
		q[ind] = I[(n*3+0)*spatial_dim+s] * mean_a[((n*maxStates+m)*channels+0)*spatial_dim+s]
					 + I[(n*3+1)*spatial_dim+s] * mean_a[((n*maxStates+m)*channels+1)*spatial_dim+s]
				   + I[(n*3+2)*spatial_dim+s] * mean_a[((n*maxStates+m)*channels+2)*spatial_dim+s]
				   + mean_b[ind];
	}

}
//---------------------------------------------
template <typename Dtype>
void GuidedCRFLayer<Dtype>::guided_filter_gpu(const int num,const int channels,const int maxStates,const int height,const int width,const Dtype *I,const Dtype * p,Dtype *output_p)
{
	const int spatial_dim=height*width;

	//******************************** prob ************************************
	box_filter_gpu(num,maxStates,height,width,radius,p,mean_p.mutable_gpu_data(),buffer_score.mutable_gpu_data());

	vector_product_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*channels*maxStates*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,maxStates,spatial_dim,I,p,Ip);//Ip = I .* p;
	
	box_filter_gpu(num,channels*maxStates,height,width,radius,Ip,mean_Ip,buffer_image_score);


	substract_vector_product_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*channels*maxStates*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,maxStates,spatial_dim,mean_Ip,mean_I.gpu_data(),mean_p.gpu_data(), cov_Ip);//cov_Ip = mean_Ip - mean_I .* mean_p;


	inv_var_I_eps_kernel_3<Dtype><<<CAFFE_GET_BLOCKS(num*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,spatial_dim,eps,var_I.mutable_gpu_data(),inv_var_I.mutable_gpu_data());//inv_var_I=inv(var_I + eps);


	div_sum_kernel_3<Dtype><<<CAFFE_GET_BLOCKS(num*maxStates*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,maxStates,spatial_dim,inv_var_I.gpu_data(),cov_Ip,a);//a = cov_Ip ./ inv_var_I;

	box_filter_gpu(num,channels*maxStates,height,width,radius,a,mean_a,buffer_image_score);

	substract_vector_matrix_product_kernel_3<Dtype><<<CAFFE_GET_BLOCKS(num*maxStates*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
  (num,channels,maxStates,spatial_dim,mean_p.gpu_data(),a,mean_I.gpu_data(),b.mutable_gpu_data());//	b = mean_p - mean_I .* a;


	box_filter_gpu(num,maxStates,height,width,radius,b.gpu_data(),mean_b.mutable_gpu_data(),buffer_score.mutable_gpu_data());

	vector_matrix_product_sum_kernel_3<Dtype><<<CAFFE_GET_BLOCKS(num*maxStates*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,maxStates,spatial_dim,mean_a,I,mean_b.gpu_data(),output_p);// q = I .* mean_a + mean_b;

}

template <typename Dtype>
void GuidedCRFLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top)
{
	const Dtype * nodePot = bottom[0]->gpu_data();
	const Dtype * imageData = bottom[1]->gpu_data();


	int num = bottom[0]->num();
	int maxStates = bottom[0]->channels();
	int channels = bottom[1]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	int spatial_dim=height*width;

	int nNodes = num*width *height;
	

	//******************************** image ************************************
	box_filter_gpu(num,channels,height,width,radius,imageData,mean_I.mutable_gpu_data(),buffer_image.mutable_gpu_data());

	vector_product_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*channels*channels*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,channels,spatial_dim,imageData,imageData,II.mutable_gpu_data());// II = I .* I;

	box_filter_gpu(num,channels*channels,height,width,radius,II.gpu_data(),mean_II.mutable_gpu_data(),buffer_image_image.mutable_gpu_data());

	substract_vector_product_kernel<Dtype><<<CAFFE_GET_BLOCKS(num*channels*channels*spatial_dim), CAFFE_CUDA_NUM_THREADS>>>
	(num,channels,channels,spatial_dim,mean_II.gpu_data(),mean_I.gpu_data(),mean_I.gpu_data(), var_I.mutable_gpu_data());//var_I = mean_II - mean_I .* mean_I;
	//-----------------------------------------------------------------------------------


	caffe_copy(tempPot.count(),nodePot,tempPot.mutable_gpu_data());
	for(int iter = 0; iter < maxIter; iter++)
	{
		softmax_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(nNodes), CAFFE_CUDA_NUM_THREADS>>>
		(maxStates,nNodes,tempPot.gpu_data(),nodeBel[iter]->mutable_gpu_data());


		guided_filter_gpu(num,channels,maxStates,height,width,imageData,nodeBel[iter]->gpu_data(),filterPot.mutable_gpu_data());
	

		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, maxStates, nNodes, maxStates,
													(Dtype)1., this->blobs_[0]->gpu_data(), filterPot.gpu_data(),
													(Dtype)0., compatPot.mutable_gpu_data());

		caffe_gpu_add_new(maxStates*nNodes,Dtype(1),nodePot,alpha,compatPot.gpu_data(),tempPot.mutable_gpu_data());
	}
	caffe_copy(top[0]->count(),tempPot.gpu_data(),top[0]->mutable_gpu_data());
}
template <typename Dtype>
void GuidedCRFLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
	int num = bottom[0]->num();
	int maxStates = bottom[0]->channels();
	int channels = bottom[1]->channels();
	int height = bottom[0]->height();
	int width = bottom[0]->width();
	int nNodes = num*width *height;

	const Dtype *top_diff = top[0]->gpu_diff();
	Dtype * bottom_diff = bottom[0]->mutable_gpu_diff();

	const Dtype * imageData = bottom[1]->gpu_data();
	

	caffe_gpu_set(filterPot.count(),Dtype(0),filterPot.mutable_gpu_diff());
	caffe_gpu_set(compatPot.count(),Dtype(0),compatPot.mutable_gpu_diff());
	caffe_gpu_set(tempPot.count(),Dtype(0),tempPot.mutable_gpu_diff());
	caffe_gpu_set(bottom[0]->count(),Dtype(0),bottom_diff);



	caffe_copy(tempPot.count(),top_diff,tempPot.mutable_gpu_diff());
	
	for(int iter = maxIter-1; iter >= 0; iter--)
	{
		caffe_gpu_add_new(maxStates*nNodes,alpha,tempPot.gpu_diff(),Dtype(0),compatPot.gpu_diff(),compatPot.mutable_gpu_diff());
		caffe_gpu_add_new(maxStates*nNodes,Dtype(1) ,tempPot.gpu_diff(),Dtype(1),bottom_diff         ,bottom_diff);


		caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, maxStates, nNodes, maxStates,
													(Dtype)1., this->blobs_[0]->gpu_data(), compatPot.gpu_diff(),
													(Dtype)0., filterPot.mutable_gpu_diff());

		guided_filter_gpu(num,channels,maxStates,height,width,imageData,filterPot.gpu_diff(),nodeBel[iter]->mutable_gpu_diff());

		softmax_backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(maxStates*nNodes), CAFFE_CUDA_NUM_THREADS>>>
		(maxStates,nNodes,nodeBel[iter]->gpu_diff(),nodeBel[iter]->gpu_data(),tempPot.mutable_gpu_diff());
	}	
	caffe_gpu_add_new(tempPot.count(),Dtype(1),tempPot.gpu_diff(),Dtype(1),bottom[0]->gpu_diff(),bottom[0]->mutable_gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(GuidedCRFLayer);
}  // namespace caffe
