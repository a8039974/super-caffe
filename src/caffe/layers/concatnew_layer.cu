#include "hip/hip_runtime.h"
#ifdef USE_OPENCV
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#endif  // USE_OPENCV
#include <algorithm>
#include <fstream>  // NOLINT(readability/streams)
#include <map>
#include <string>
#include <utility>
#include <vector>
#include <iostream>
#include "boost/filesystem.hpp"
#include "boost/foreach.hpp"

#include "caffe/layers/concatnew_layer.hpp"
#include "caffe/util/math_functions.hpp"
using namespace std;
namespace caffe {

template <typename Dtype>
__global__ void ConcatNew(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_concats, const int concat_size,
    const int top_concat_axis, const int bottom_concat_axis,
    const int offset_concat_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_concat_size = concat_size * bottom_concat_axis;
    const int concat_num = index / total_concat_size;
    const int concat_index = index % total_concat_size;
    const int top_index = concat_index +
        (concat_num * top_concat_axis + offset_concat_axis) * concat_size;
    if (forward) {
      out_data[top_index] = in_data[index];
    } else {
      out_data[index] = in_data[top_index];
    }
  }
}

template <typename Dtype>
void ConcatNewLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  if (visualize_) {
#ifdef USE_OPENCV
    vector<cv::Mat> cv_imgs,cv_seg,cv_imgseg;
    this->data_transformer_->TransformInv(bottom[0], &cv_imgs);
    this->data_transformer_->TransformInv(bottom[1], &cv_seg);
    this->data_transformer_->TransformInv(bottom[3], &cv_imgseg);
    //vector<cv::Mat> new_imgs = AddChannels(cv_imgs, cv_seg);
    vector<cv::Scalar> colors = GetColors(label_to_display_name_.size());
    VisualizeBBox(cv_imgs, cv_seg, cv_imgseg, bottom[2], visualize_threshold_, colors,
        label_to_display_name_, save_file_);
#endif  // USE_OPENCV
  }
}

template <typename Dtype>
void ConcatNewLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    NOT_IMPLEMENTED;
}

INSTANTIATE_LAYER_GPU_FUNCS(ConcatNewLayer);

}  // namespace caffe
