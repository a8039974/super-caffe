#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/sliceseg_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Sliceseg(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_slicesegs, const int sliceseg_size,
    const int bottom_sliceseg_axis, const int top_sliceseg_axis,
    const int offset_sliceseg_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_sliceseg_size = sliceseg_size * top_sliceseg_axis;
    const int sliceseg_num = index / total_sliceseg_size;
    const int sliceseg_index = index % total_sliceseg_size;
    const int bottom_index = sliceseg_index +
        (sliceseg_num * bottom_sliceseg_axis + offset_sliceseg_axis) * sliceseg_size;
    if (forward) {
      out_data[index] = in_data[bottom_index];
    } else {
      out_data[bottom_index] = in_data[index];
    }
  }
}

template <typename Dtype>
void SlicesegLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  //if (top.size() == 1) { return; }
  int offset_sliceseg_axis = 0;
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const int bottom_sliceseg_axis = bottom[0]->shape(sliceseg_axis_);
  const bool kForward = true;
  for (int i = 0; i < 1; ++i) {
    Dtype* top_data = top[i]->mutable_gpu_data();
    const int top_sliceseg_axis = top[i]->shape(sliceseg_axis_);
    const int top_sliceseg_size = top_sliceseg_axis * sliceseg_size_;
    const int nthreads = top_sliceseg_size * num_slicesegs_;
    Sliceseg<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_slicesegs_, sliceseg_size_,
        bottom_sliceseg_axis, top_sliceseg_axis, offset_sliceseg_axis, top_data);
    offset_sliceseg_axis += top_sliceseg_axis;
  }
}

template <typename Dtype>
void SlicesegLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    NOT_IMPLEMENTED;
}

INSTANTIATE_LAYER_GPU_FUNCS(SlicesegLayer);

}  // namespace caffe
